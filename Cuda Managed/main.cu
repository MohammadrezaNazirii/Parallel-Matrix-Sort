#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef USE_INT
    typedef int matrix_type;
    #define FORMAT_SPECIFIER "%d"
#elif defined(USE_FLOAT)
    typedef float matrix_type;
    #define FORMAT_SPECIFIER "%f"
#elif defined(USE_DOUBLE)
    typedef double matrix_type;
    #define FORMAT_SPECIFIER "%lf"
#else
    #error "Please define USE_INT, USE_FLOAT, or USE_DOUBLE."
#endif

#define CHUNK_SIZE 1024


int N;
int BLOCK_SIZE;


// Function to read a matrix from a file
void read_matrix(const char* filename, matrix_type** matrix, int size) {
    FILE *fin = fopen(filename, "r");
    if (fin == NULL) {
        perror("Error opening input file");
        exit(EXIT_FAILURE);
    }

    // Reading the matrix elements
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (fscanf(fin, FORMAT_SPECIFIER, &matrix[i][j]) != 1) {
                perror("Error reading matrix from file");
                exit(EXIT_FAILURE);
            }
        }
    }

    fclose(fin);
}

// Function to write a matrix to a file
void write_matrix(const char* filename, matrix_type** matrix, int size) {
    FILE *fout = fopen(filename, "w");
    if (fout == NULL) {
        perror("Error opening output file");
        exit(EXIT_FAILURE);
    }

    // Writing the matrix elements
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            fprintf(fout, FORMAT_SPECIFIER " ", matrix[i][j]);
        }
        fprintf(fout, "\n");  // End each row with a newline
    }

    fclose(fout);
}




// CUDA kernel to sort rows within a chunk
__global__ void sortRows(matrix_type* matrix, int numRows, int numCols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows) {
        for (int i = 0; i < numCols - 1; ++i) {
            for (int j = 0; j < numCols - i - 1; ++j) {
                int idx = row * numCols + j;
                if (matrix[idx] > matrix[idx + 1]) {
                    matrix_type temp = matrix[idx];
                    matrix[idx] = matrix[idx + 1];
                    matrix[idx + 1] = temp;
                }
            }
        }
    }
}

// CUDA kernel to sort columns within a chunk
__global__ void sortColumns(matrix_type* matrix, int numRows, int numCols, int* changesMade) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < numCols) {
        for (int i = 0; i < numRows - 1; ++i) {
            for (int j = 0; j < numRows - i - 1; ++j) {
                int idx = j * numCols + col;
                if (matrix[idx] > matrix[(j + 1) * numCols + col]) {
                    matrix_type temp = matrix[idx];
                    matrix[idx] = matrix[(j + 1) * numCols + col];
                    matrix[(j + 1) * numCols + col] = temp;

                    // Use atomic operation to set the flag
                    atomicExch(changesMade, 1);  // Set to 1 (true)
                }
            }
        }
    }
}



float total_time = 0;
void processMatrixChunks(matrix_type** matrix, int numRows, int numCols, const char* filename) {
    int numChunks = numRows / CHUNK_SIZE;
    int* d_changesMade;

    // Allocate unified memory for changesMade as an int
    hipMallocManaged(&d_changesMade, sizeof(int));
    int temp = numRows/1024;
    printf("%d", temp);

    do {
        *d_changesMade = 0; // Reset the flag to 0 (false)
        temp--;

        // Process row chunks
        for (int chunk = 0; chunk < numChunks; ++chunk) {
            int numBlocks = (CHUNK_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;

            // Timing
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);

            // Sort rows in the chunk
            sortRows<<<numBlocks, BLOCK_SIZE>>>(matrix[chunk], CHUNK_SIZE, numCols);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Row sorting time: %lf ms\n", milliseconds);
            total_time += milliseconds;
        }

        // Process column chunks for merging
        int chunkCols = numCols / CHUNK_SIZE;
        for (int chunk = 0; chunk < chunkCols; ++chunk) {
            int numBlocks = (CHUNK_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;

            // Timing
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);

            // Sort columns in the chunk
            sortColumns<<<numBlocks, BLOCK_SIZE>>>(matrix[chunk], numRows, CHUNK_SIZE, d_changesMade);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Column sorting time: %lf ms\n", milliseconds);
            total_time += milliseconds;
        }

        hipDeviceSynchronize(); // Ensure all device computations are done
        if(temp == 0){
            break;
        }
    } while (*d_changesMade != 0); // Check if changes were made

    write_matrix(filename, matrix, numRows);

    // Free the unified memory for changesMade
    hipFree(d_changesMade);
}


int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <input_matrix_file> <output_matrix_file>\n", argv[0]);
        return EXIT_FAILURE;
    }



    const char* input_file = argv[1];
    const char* output_file = argv[2];

    // std::string output_file = output_fil;

    BLOCK_SIZE = atoi(argv[3]);


     if (sscanf(input_file, "matrix_%*[^_]_%dx%d", &N, &N) != 2) {
        printf("Error: Could not determine matrix size from filename. Expected format is matrix_TYPE_SIZExSIZE.txt\n");
        return EXIT_FAILURE;
    }


   matrix_type** matrix;
    // Allocate memory for the 2D array

     hipError_t err = hipMallocManaged(&matrix, N * sizeof(matrix_type*));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for row pointers: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    // Step 2: Allocate memory for each row
    for (int i = 0; i < N; ++i) {
        err = hipMallocManaged(&matrix[i], N * sizeof(matrix_type));
        if (err != hipSuccess) {
            printf("CUDA malloc failed for row %d: %s\n", i, hipGetErrorString(err));
            return EXIT_FAILURE;
        }
    }

    hipDeviceSynchronize();

    read_matrix(input_file, matrix, N);



    processMatrixChunks(matrix, N, N, output_file);

  hipFree(matrix);

    printf("total: %lf\n", total_time);

    return 0;
}



