#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef USE_INT
    typedef int matrix_type;
    #define FORMAT_SPECIFIER "%d"
#elif defined(USE_FLOAT)
    typedef float matrix_type;
    #define FORMAT_SPECIFIER "%f"
#elif defined(USE_DOUBLE)
    typedef double matrix_type;
    #define FORMAT_SPECIFIER "%lf"
#else
    #error "Please define USE_INT, USE_FLOAT, or USE_DOUBLE."
#endif

#define CHUNK_SIZE 1024


int N;
int BLOCK_SIZE;


// Function to read a matrix from a file
void read_matrix(const char* filename, matrix_type** matrix, int size) {
    FILE *fin = fopen(filename, "r");
    if (fin == NULL) {
        perror("Error opening input file");
        exit(EXIT_FAILURE);
    }

    // Reading the matrix elements
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (fscanf(fin, FORMAT_SPECIFIER, &matrix[i][j]) != 1) {
                perror("Error reading matrix from file");
                exit(EXIT_FAILURE);
            }
        }
    }

    fclose(fin);
}

// Function to write a matrix to a file
void write_matrix(const char* filename, matrix_type** matrix, int size) {
    FILE *fout = fopen(filename, "w");
    if (fout == NULL) {
        perror("Error opening output file");
        exit(EXIT_FAILURE);
    }

    // Writing the matrix elements
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            fprintf(fout, FORMAT_SPECIFIER " ", matrix[i][j]);
        }
        fprintf(fout, "\n");  // End each row with a newline
    }

    fclose(fout);
}




// CUDA kernel to sort rows within a chunk
__global__ void sortRows(matrix_type* matrix, int numRows, int numCols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows) {
        for (int i = 0; i < numCols - 1; ++i) {
            for (int j = 0; j < numCols - i - 1; ++j) {
                int idx = row * numCols + j;
                if (matrix[idx] > matrix[idx + 1]) {
                    matrix_type temp = matrix[idx];
                    matrix[idx] = matrix[idx + 1];
                    matrix[idx + 1] = temp;
                }
            }
        }
    }
}

// CUDA kernel to sort columns within a chunk
__global__ void sortColumns(matrix_type* matrix, int numRows, int numCols, bool* changesMade) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < numCols) {
        bool localChanges = false;
        for (int i = 0; i < numRows - 1; ++i) {
            for (int j = 0; j < numRows - i - 1; ++j) {
                int idx = j * numCols + col;
                if (matrix[idx] > matrix[(j + 1) * numCols + col]) {
                    matrix_type temp = matrix[idx];
                    matrix[idx] = matrix[(j + 1) * numCols + col];
                    matrix[(j + 1) * numCols + col] = temp;
                    localChanges = true;
                }
            }
        }
        if (localChanges) {
            *changesMade = true;
        }
    }
}

float total_time = 0;
void processMatrixChunks(matrix_type** matrix, int numRows, int numCols, const char* filename) {
    int numChunks = numRows / CHUNK_SIZE;
    std::vector<matrix_type*> d_matrix(numChunks);

    std::vector<hipStream_t> streams(numChunks);
    std::vector<bool*> d_changesMade(numChunks);

    // Allocate device memory and create streams for each chunk
    for (int i = 0; i < numChunks; ++i) {
        hipMalloc(&d_matrix[i], CHUNK_SIZE * numCols * sizeof(matrix_type));
        hipMalloc(&d_changesMade[i], sizeof(bool));
        hipStreamCreate(&streams[i]);
    }
 bool changes;
        do {
            changes = false;
    // Process row chunks
    for (int chunk = 0; chunk < numChunks; ++chunk) {
        // Copy chunk to device
        for (int row = 0; row < CHUNK_SIZE; ++row) {
            hipMemcpyAsync(&d_matrix[chunk][row * numCols], matrix[chunk * CHUNK_SIZE + row], numCols * sizeof(matrix_type),
                            hipMemcpyHostToDevice, streams[chunk]);
        }

        // Sort rows in the chunk
        int numBlocks = (CHUNK_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;



        // Start and stop events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Record start event
        hipEventRecord(start, 0);



        sortRows<<<numBlocks, BLOCK_SIZE, 0, streams[chunk]>>>(d_matrix[chunk], CHUNK_SIZE, numCols);
        // hipStreamSynchronize(streams[chunk]);

        // Record stop event
        hipEventRecord(stop, 0);

        // Synchronize to ensure all work is done
        hipEventSynchronize(stop);

        // Calculate and print elapsed time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // printf("this time: %lf\n", milliseconds);


        total_time += milliseconds;



        // Copy chunk back to host
        for (int row = 0; row < CHUNK_SIZE; ++row) {
            hipMemcpyAsync(matrix[chunk * CHUNK_SIZE + row], &d_matrix[chunk][row * numCols], numCols * sizeof(matrix_type),
                            hipMemcpyDeviceToHost, streams[chunk]);
        }
    }

    // Process column chunks for merging
    int chunkCols = numCols / CHUNK_SIZE;
    for (int chunk = 0; chunk < chunkCols; ++chunk) {


            hipMemcpyAsync(d_changesMade[chunk], &changes, sizeof(bool), hipMemcpyHostToDevice, streams[chunk]);

            // Copy column chunk to device
            for (int row = 0; row < numRows; ++row) {
                hipMemcpyAsync(&d_matrix[chunk][row * CHUNK_SIZE], &matrix[row][chunk * CHUNK_SIZE],
                                CHUNK_SIZE * sizeof(matrix_type), hipMemcpyHostToDevice, streams[chunk]);
            }

            // Start and stop events
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            // Record start event
            hipEventRecord(start, 0);


            // Sort columns in the chunk
            int numBlocks = (CHUNK_SIZE + BLOCK_SIZE -1 ) / BLOCK_SIZE;
            sortColumns<<<numBlocks, BLOCK_SIZE, 0, streams[chunk]>>>(d_matrix[chunk], numRows, CHUNK_SIZE, d_changesMade[chunk]);
            // hipStreamSynchronize(streams[chunk]);

            // Record stop event
            hipEventRecord(stop, 0);

            // Synchronize to ensure all work is done
            hipEventSynchronize(stop);

            // Calculate and print elapsed time
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            total_time += milliseconds;

            // Copy column chunk back to host
            for (int row = 0; row < numRows; ++row) {
                hipMemcpyAsync(&matrix[row][chunk * CHUNK_SIZE], &d_matrix[chunk][row * CHUNK_SIZE],
                                CHUNK_SIZE * sizeof(matrix_type), hipMemcpyDeviceToHost, streams[chunk]);
            }

            // Check if any changes were made
            hipMemcpy(&changes, d_changesMade[chunk], sizeof(bool), hipMemcpyDeviceToHost);
        }
    }while (changes); // Repeat if changes were made

    // Free resources
    for (int i = 0; i < numChunks; ++i) {
        hipFree(d_matrix[i]);
        hipFree(d_changesMade[i]);
        hipStreamDestroy(streams[i]);
    }



    write_matrix(filename, matrix, numRows);
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <input_matrix_file> <output_matrix_file>\n", argv[0]);
        return EXIT_FAILURE;
    }



    const char* input_file = argv[1];
    const char* output_file = argv[2];
    // std::string output_file = output_fil;

    BLOCK_SIZE = atoi(argv[3]);


     if (sscanf(input_file, "matrix_%*[^_]_%dx%d", &N, &N) != 2) {
        printf("Error: Could not determine matrix size from filename. Expected format is matrix_TYPE_SIZExSIZE.txt\n");
        return EXIT_FAILURE;
    }


    matrix_type** matrix = new matrix_type*[N];
    for (int i = 0; i < N; ++i) {
        matrix[i] = new matrix_type[N];
    }


    read_matrix(input_file, matrix, N);

    //
    // Process matrix chunks
    processMatrixChunks(matrix, N, N, output_file);

    // Free dynamically allocated memory
    for (int i = 0; i < N; ++i) {
        delete[] matrix[i];

    }
    delete[] matrix;

    printf("%lf\n", total_time);

    return 0;
}



